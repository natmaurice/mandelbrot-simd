#include "hip/hip_runtime.h"
#include "mandelbrot-cuda.h"
#include <bits/time.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>


__global__
void mandelbrot_kernel(int32_t* mat, int nrl, int nrh, int ncl, int nch, int stride,
		       float minx, float miny, float maxx, float maxy, int max_iterations) {


    int width = nch - ncl + 1;
    int height = nrh - nrl + 1;

    float stepx = (maxx - minx) / width;
    float stepy = (maxy - miny) / height;
    
    for (int ri = threadIdx.y + blockDim.y * blockIdx.y; ri < height; ri += blockDim.y * gridDim.y) {
	int32_t* line = mat + ri * stride;
	const int i = ri - nrl;
	const float y0 = miny + i * stepy;
	
	for (int rj = threadIdx.x + blockDim.x * blockIdx.x; rj < width; rj += blockDim.x * gridDim.x) {
	    const int j = rj - ncl;

	    int it = 0;

	    const float x0 = minx + j * stepx;
	    float x = 0, y = 0;
	    float x2 = 0, y2 = 0;

	    int final_it = max_iterations;
	    int mask = 0xffffffff;
	    
	    for (it = 0; it < max_iterations; it++) {
		
		y = 2 * x * y + y0;
		x = x2 - y2 + x0;
		x2 = x * x;
		y2 = y * y;
		
		if (x2 + y2 > 2*2) {
		    // Don't modify final_it if this condition has already been encountered
		    final_it = (it & mask) | (final_it & (~mask));
		    mask = 0x0;
		}
	    }
	    line[j] = final_it;
	}
    }
}

__global__
void mandelbrot_kernel_f16(int32_t* mat, int nrl, int nrh, int ncl, int nch, int stride,
			   float minx, float miny, float maxx, float maxy, int max_iterations) {


    int width = nch - ncl + 1;
    int height = nrh - nrl + 1;

    float stepx = (maxx - minx) / width;
    float stepy = (maxy - miny) / height;
    
    for (int ri = threadIdx.y + blockDim.y * blockIdx.y; ri < height; ri += blockDim.y * gridDim.y) {
	int32_t* line = mat + ri * stride;
	const int i = ri - nrl;
	const float y0 = miny + i * stepy;
	
	for (int rj = threadIdx.x + blockDim.x * blockIdx.x; rj < width; rj += blockDim.x * gridDim.x) {
	    const int j = rj - ncl;

	    int it = 0;

	    const float x0 = minx + j * stepx;
	    float x = 0, y = 0;
	    float x2 = 0, y2 = 0;

	    int final_it = max_iterations;
	    int mask = 0xffffffff;
	    
	    for (it = 0; it < max_iterations; it++) {
		
		y = 2 * x * y + y0;
		x = x2 - y2 + x0;
		x2 = x * x;
		y2 = y * y;
		
		if (x2 + y2 > 2*2) {
		    // Don't modify final_it if this condition has already been encountered
		    final_it = (it & mask) | (final_it & (~mask));
		    mask = 0x0;
		}
	    }
	    line[j] = final_it;
	}
    }
}



__global__
void mandelbrot_kernel_v2(int32_t* mat, int nrl, int nrh, int ncl, int nch, int stride,
			  float minx, float miny, float maxx, float maxy, int max_iterations) {


    int width = nch - ncl + 1;
    int height = nrh - nrl + 1;

    float stepx = (maxx - minx) / width;
    float stepy = (maxy - miny) / height;
    
    for (int ri = threadIdx.y + blockDim.y * blockIdx.y; ri < height; ri += blockDim.y * gridDim.y) {
	int32_t* line = mat + ri * stride;
	const int i = ri - nrl;
	const float y0 = miny + i * stepy;
	
	for (int rj = threadIdx.x + blockDim.x * blockIdx.x; rj < width; rj += blockDim.x * gridDim.x) {
	    const int j = rj - ncl;

	    int it = 0;

	    const float x0 = minx + j * stepx;
	    float x = 0, y = 0;
	    float x2 = 0, y2 = 0;

	    int final_it = max_iterations;
	    int mask = 0xffffffff;
	    
	    for (it = 0; it < max_iterations; it++) {
		
		y = 2 * x * y + y0;
		x = x2 - y2 + x0;
		x2 = x * x;
		y2 = y * y;
		
		if (x2 + y2 > 2*2) {
		    break;
		}
	    }
	    line[j] = it;
	}
    }
}


__global__
void mandelbrot_kernel_lu2(int32_t* mat, int nrl, int nrh, int ncl, int nch, int stride,
			   float minx, float miny, float maxx, float maxy, int max_iterations) {


    int width = nch - ncl + 1;
    int height = nrh - nrl + 1;

    float stepx = (maxx - minx) / width;
    float stepy = (maxy - miny) / height;

    constexpr int ELEM_PER_IT = 2;
    
    for (int ri = threadIdx.y + blockDim.y * blockIdx.y; ri < height; ri += blockDim.y * gridDim.y) {
	int32_t* line = mat + ri * stride;
	const int i = ri - nrl;
	const float vy0 = miny + i * stepy;
	
	for (int rj = (threadIdx.x + blockDim.x * blockIdx.x) * ELEM_PER_IT;
	     rj < width;
	     rj += (blockDim.x * gridDim.x) * ELEM_PER_IT) {

	    
	    const int j = rj - ncl;

	    int it = 0;
	    int final_it0 = max_iterations, final_it1 = max_iterations;
	    
	    const float vx0 = minx + j * stepx;
	    const float vx1 = minx + ((j + 1) * stepx);
	    
	    float x0 = 0, y0 = 0;
	    float xb0 = 0, yb0 = 0;
	    float x1 = 0, y1 = 0;
	    float xb1 = 0, yb1 = 0;

	    int mask0, mask1;
	    mask0 = mask1 = 0xffffffff;
	    
	    for (it = 0; it < max_iterations; it++) {
		
		y0 = 2 * x0 * y0 + vy0;
		y1 = 2 * x1 * y1 + vy0;
		
		x0 = xb0 - yb0 + vx0;
		x1 = xb1 - yb1 + vx1;

		xb0 = x0 * x0;
		xb1 = x1 * x1;

		yb0 = y0 * y0;
		yb1 = y1 * y1;
		
		if (xb0 + yb0 > 2*2) {
		    // Don't modify final_it if this condition has already been encountered
		    final_it0 = (it & mask0) | (final_it0 & (~mask0));
		    mask0 = 0x0;
		}
		if (xb1 + xb1 > 2*2) {
		    final_it1 = (it & mask1) | (final_it1 & (~mask1));
		    mask1 = 0x0;
		}
	    }
	    line[j    ] = final_it0;
	    line[j + 1] = final_it1;
	}
    }
}




struct Mandelbrot_V0 {
    static void Execute(dim3 nblocks, dim3 threadsPerBlock, 
			    int32_t* dev_mat, int nrl, int nrh, int ncl, int nch, int stride,
			    float minx, float miny, float maxx, float maxy, int max_iterations) {
    
	mandelbrot_kernel<<<nblocks, threadsPerBlock>>>(dev_mat, nrl, nrh, ncl, nch, stride, minx, miny, maxx, maxy, max_iterations);
    }
};


struct Mandelbrot_V2 {
    static void Execute(dim3 nblocks, dim3 threadsPerBlock, 
			    int32_t* dev_mat, int nrl, int nrh, int ncl, int nch, int stride,
			    float minx, float miny, float maxx, float maxy, int max_iterations) {
    
	mandelbrot_kernel_v2<<<nblocks, threadsPerBlock>>>(dev_mat, nrl, nrh, ncl, nch, stride, minx, miny, maxx, maxy, max_iterations);
    }
};


struct Mandelbrot_LU2 {
    static void Execute(dim3 nblocks, dim3 threadsPerBlock, 
			    int32_t* dev_mat, int nrl, int nrh, int ncl, int nch, int stride,
			    float minx, float miny, float maxx, float maxy, int max_iterations) {
    
	mandelbrot_kernel_lu2<<<nblocks, threadsPerBlock>>>(dev_mat, nrl, nrh, ncl, nch, stride, minx, miny, maxx, maxy, max_iterations);
    }
};


template <class Kernel>    
void mandelbrot_generic(int32_t** mat, int nrl, int nrh, int ncl, int nch,
			float minx, float miny, float maxx, float maxy, int max_iterations) {
    
    int height = nrh - nrl + 1;
    int width = nch - ncl + 1;

    
    dim3 threadsPerBlock(16, 16);
    dim3 nblocks(width / threadsPerBlock.x, height / threadsPerBlock.y);
    //dim3 nblocks(width / threadPerBlock.x, height / threadPerBlock.y);
    
    int stride = width;
    if (height > 1) {
	stride = mat[1] - mat[0];
    }

    int32_t* dev_mat;
    
    hipMalloc((void**)&dev_mat, height*stride*sizeof(int32_t));


    struct timespec ts_start, ts_end;
    clock_gettime(CLOCK_MONOTONIC_RAW, &ts_start);
    
    Kernel::Execute(nblocks, threadsPerBlock, dev_mat, nrl, nrh, ncl, nch, stride, minx, miny, maxx, maxy, max_iterations);
    hipDeviceSynchronize();
    
    clock_gettime(CLOCK_MONOTONIC_RAW, &ts_end);

    double elapsed_ms = (ts_end.tv_sec - ts_start.tv_sec) * 1000 + (ts_end.tv_nsec - ts_start.tv_nsec) / 1e6;
    
    
    hipMemcpy(mat[0], dev_mat, height*stride*sizeof(int32_t), hipMemcpyDeviceToHost);

    long pixel_count = width * height;
    
    float pixel_per_s = pixel_count / (elapsed_ms / 1000.0);
    float gpixels = pixel_per_s / 1e9;
    
    printf("Elapsed GPU time = %f\n", elapsed_ms);
    printf("Bandwidth = %f GPix/s\n", gpixels);


    
    hipFree(dev_mat);
}

void mandelbrot_cuda(int32_t **mat, int nrl, int nrh, int ncl, int nch,
                     float minx, float miny, float maxx, float maxy,
                     int max_iterations) {
    
    mandelbrot_generic<Mandelbrot_V0>(mat, nrl, nrh, ncl, nch, minx, miny, maxx, maxy, max_iterations);
}


void mandelbrot_cuda_v2(int32_t **mat, int nrl, int nrh, int ncl, int nch,
                     float minx, float miny, float maxx, float maxy,
                     int max_iterations) {
    
    mandelbrot_generic<Mandelbrot_V2>(mat, nrl, nrh, ncl, nch, minx, miny, maxx, maxy, max_iterations);
}


void mandelbrot_cuda_lu2(int32_t **mat, int nrl, int nrh, int ncl, int nch,
                     float minx, float miny, float maxx, float maxy,
                     int max_iterations) {
    mandelbrot_generic<Mandelbrot_LU2>(mat, nrl, nrh, ncl, nch, minx, miny, maxx, maxy, max_iterations);
}